#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "IVFBase.cuh"
#include "../GpuResources.h"
#include "InvertedListAppend.cuh"
#include "../utils/DeviceDefs.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/HostTensor.cuh"
#include <limits>
#include <thrust/host_vector.h>
#include <unordered_map>

namespace faiss { namespace gpu {

IVFBase::IVFBase(GpuResources* resources,
                 IndicesOptions indicesOptions,
                 MemorySpace space,
                 int nlist) :
    resources_(resources),
    indicesOptions_(indicesOptions),
    space_(space),
    numLists_(nlist),
    maxListLength_(0) {
  reset();
}

IVFBase::~IVFBase() {
}

void
IVFBase::reserveMemory(size_t numVecs) {
  size_t vecsPerList = numVecs / deviceListIndices_.size();
  if (vecsPerList < 1) {
    return;
  }

  auto stream = resources_->getDefaultStreamCurrentDevice();

  if ((indicesOptions_ == INDICES_32_BIT) ||
      (indicesOptions_ == INDICES_64_BIT)) {
    // Reserve for index lists as well
    size_t bytesPerIndexList = vecsPerList *
      (indicesOptions_ == INDICES_32_BIT ? sizeof(int) : sizeof(long));

    for (auto& list : deviceListIndices_) {
      list->reserve(bytesPerIndexList, stream);
    }
  }

  // Update device info for all lists, since the base pointers may
  // have changed
  updateDeviceListInfo_(stream);
}

void
IVFBase::reset() {
  deviceListIndices_.clear();
  deviceListIndexPointers_.clear();
  deviceListLengths_.clear();

  for (size_t i = 0; i < numLists_; ++i) {
    deviceListIndices_.emplace_back(
      std::unique_ptr<DeviceVector<unsigned char>>(
        new DeviceVector<unsigned char>(space_)));
  }

  deviceListIndexPointers_.resize(numLists_, nullptr);
  deviceListLengths_.resize(numLists_, 0);
  maxListLength_ = 0;
}

size_t
IVFBase::reclaimMemory() {
  // Reclaim all unused memory exactly
  return reclaimMemory_(true);
}

size_t
IVFBase::reclaimMemory_(bool exact) {
  auto stream = resources_->getDefaultStreamCurrentDevice();

  size_t totalReclaimed = 0;

  for (int i = 0; i < deviceListIndices_.size(); ++i) {
    auto& indices = deviceListIndices_[i];
    totalReclaimed += indices->reclaim(exact, stream);

    deviceListIndexPointers_[i] = indices->data();
  }

  // Update device info for all lists, since the base pointers may
  // have changed
  updateDeviceListInfo_(stream);

  return totalReclaimed;
}

void
IVFBase::updateDeviceListInfo_(hipStream_t stream) {
  std::vector<int> listIds(deviceListIndices_.size());
  for (int i = 0; i < deviceListIndices_.size(); ++i) {
    listIds[i] = i;
  }

  updateDeviceListInfo_(listIds, stream);
}

void
IVFBase::updateDeviceListInfo_(const std::vector<int>& listIds,
                               hipStream_t stream) {
  auto& mem = resources_->getMemoryManagerCurrentDevice();

  HostTensor<int, 1, true>
    hostListsToUpdate({(int) listIds.size()});
  HostTensor<int, 1, true>
    hostNewListLength({(int) listIds.size()});
  HostTensor<void*, 1, true>
    hostNewIndexPointers({(int) listIds.size()});

  size_t indiceSize =
    (indicesOptions_ == INDICES_32_BIT ? sizeof(int) : sizeof(long));

  for (int i = 0; i < listIds.size(); ++i) {
    auto listId = listIds[i];
    auto& indices = deviceListIndices_[listId];

    hostListsToUpdate[i] = listId;
    hostNewListLength[i] = indices->size() / indiceSize;
    hostNewIndexPointers[i] = indices->data();
  }

  // Copy the above update sets to the GPU
  DeviceTensor<int, 1, true> listsToUpdate(
    mem, hostListsToUpdate, stream);
  DeviceTensor<int, 1, true> newListLength(
    mem,  hostNewListLength, stream);
  DeviceTensor<void*, 1, true> newIndexPointers(
    mem, hostNewIndexPointers, stream);

  // Update all pointers to the lists on the device that may have
  // changed
  runUpdateListPointers(listsToUpdate,
                        newListLength,
                        newIndexPointers,
                        deviceListLengths_,
                        deviceListIndexPointers_,
                        stream);
}

size_t
IVFBase::getNumLists() const {
  return numLists_;
}

int
IVFBase::getListLength(int listId) const {
  FAISS_ASSERT(listId < deviceListLengths_.size());

  return deviceListLengths_[listId];
}

std::vector<long>
IVFBase::getListIndices(int listId) const {
  FAISS_ASSERT(listId < numLists_);
  FAISS_ASSERT(listId < deviceListIndices_.size());

  if (indicesOptions_ == INDICES_32_BIT) {
    auto intInd = deviceListIndices_[listId]->copyToHost<int>(
      resources_->getDefaultStreamCurrentDevice());

    std::vector<long> out(intInd.size());
    for (size_t i = 0; i < intInd.size(); ++i) {
      out[i] = (long) intInd[i];
    }

    return out;
  } else if (indicesOptions_ == INDICES_64_BIT) {

    return deviceListIndices_[listId]->copyToHost<long>(
      resources_->getDefaultStreamCurrentDevice());
  } else {
    // unhandled indices type (includes INDICES_IVF)
    FAISS_ASSERT(false);
    return std::vector<long>();
  }
}

void
IVFBase::addIndicesFromCpu_(int listId,
                            const long* indices,
                            size_t numVecs) {
  auto stream = resources_->getDefaultStreamCurrentDevice();

  auto& listIndices = deviceListIndices_[listId];
  auto prevIndicesData = listIndices->data();

  if (indicesOptions_ == INDICES_32_BIT) {
    // Make sure that all indices are in bounds
    std::vector<int> indices32(numVecs);
    for (size_t i = 0; i < numVecs; ++i) {
      auto ind = indices[i];
      FAISS_ASSERT(ind <= (long) std::numeric_limits<int>::max());
      indices32[i] = (int) ind;
    }

    listIndices->append((unsigned char*) indices32.data(),
                        numVecs * sizeof(int),
                        stream,
                        true /* exact reserved size */);
  } else if (indicesOptions_ == INDICES_64_BIT) {
    listIndices->append((unsigned char*) indices,
                        numVecs * sizeof(long),
                        stream,
                        true /* exact reserved size */);
  } else {
    // indices are not stored
    FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
  }

  if (prevIndicesData != listIndices->data()) {
    deviceListIndexPointers_[listId] = listIndices->data();
  }
}

} } // namespace
