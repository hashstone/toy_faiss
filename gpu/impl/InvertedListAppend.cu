#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "InvertedListAppend.cuh"
#include "../utils/FaissAssert.h"
#include "../utils/Float16.cuh"
#include "../utils/DeviceUtils.h"
#include "../utils/Tensor.cuh"
#include "../utils/StaticUtils.h"

#include <iostream>

namespace faiss { namespace gpu {

__global__ void
runUpdateListPointers(Tensor<int, 1, true> listIds,
                      Tensor<int, 1, true> newListLength,
                      Tensor<void*, 1, true> newIndexPointers,
                      int* listLengths,
                      void** listIndices) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= listIds.getSize(0)) {
    return;
  }

  int listId = listIds[index];
  listLengths[listId] = newListLength[index];
  listIndices[listId] = newIndexPointers[index];
}

void
runUpdateListPointers(Tensor<int, 1, true>& listIds,
                      Tensor<int, 1, true>& newListLength,
                      Tensor<void*, 1, true>& newIndexPointers,
                      thrust::device_vector<int>& listLengths,
                      thrust::device_vector<void*>& listIndices,
                      hipStream_t stream) {
  int numThreads = std::min(listIds.getSize(0), getMaxThreadsCurrentDevice());
  int numBlocks = utils::divUp(listIds.getSize(0), numThreads);

  dim3 grid(numBlocks);
  dim3 block(numThreads);

  runUpdateListPointers<<<grid, block, 0, stream>>>(
    listIds, newListLength, newIndexPointers,
    listLengths.data().get(),
    listIndices.data().get());

  CUDA_TEST_ERROR();
}

template <IndicesOptions Opt>
__global__ void
ivfpqInvertedListAppend(Tensor<int, 1, true> listIds,
                        Tensor<int, 1, true> listOffset,
                        Tensor<long, 1, true> indices,
                        void** listIndices) {
  int encodingToAdd = blockIdx.x * blockDim.x + threadIdx.x;

  if (encodingToAdd >= listIds.getSize(0)) {
    return;
  }

  int listId = listIds[encodingToAdd];
  int offset = listOffset[encodingToAdd];

  // Add vector could be invalid (contains NaNs etc)
  if (listId == -1 || offset == -1) {
    return;
  }

  long index = indices[encodingToAdd];

  if (Opt == INDICES_32_BIT) {
    // FIXME: there could be overflow here, but where should we check this?
    ((int*) listIndices[listId])[offset] = (int) index;
  } else if (Opt == INDICES_64_BIT) {
    ((long*) listIndices[listId])[offset] = (long) index;
  } else {
    // INDICES_CPU or INDICES_IVF; no indices are being stored
  }
}

void
runIVFIDInvertedListAppend(Tensor<int, 1, true>& listIds,
                           Tensor<int, 1, true>& listOffset,
                           Tensor<long, 1, true>& indices,
                           thrust::device_vector<void*>& listIndices,
                           IndicesOptions indicesOptions,
                           hipStream_t stream) {
  int numThreads = std::min(listIds.getSize(0), getMaxThreadsCurrentDevice());
  int numBlocks = utils::divUp(listIds.getSize(0), numThreads);

  dim3 grid(numBlocks);
  dim3 block(numThreads);

#define RUN_APPEND(IND)                                         \
  do {                                                          \
    ivfpqInvertedListAppend<IND><<<grid, block, 0, stream>>>(   \
      listIds, listOffset, indices,                  \
      listIndices.data().get());                                \
  } while (0)

  if ((indicesOptions == INDICES_CPU) || (indicesOptions == INDICES_IVF)) {
    // no need to maintain indices on the GPU
    RUN_APPEND(INDICES_IVF);
  } else if (indicesOptions == INDICES_32_BIT) {
    RUN_APPEND(INDICES_32_BIT);
  } else if (indicesOptions == INDICES_64_BIT) {
    RUN_APPEND(INDICES_64_BIT);
  } else {
    // unknown index storage type
    FAISS_ASSERT(false);
  }

  CUDA_TEST_ERROR();

#undef RUN_APPEND
}

template <IndicesOptions Opt>
__global__ void
ivfpqInvertedListFind(long id,
                      int nlist,
                      void** listIndices,
                      int* listLengths,
                      Tensor<int, 1, true> offset) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx >= nlist) {
    return;
  }

  // FIXME: loop in kernel? 
  offset[idx] = -1;
  int total = listLengths[idx];
  if (Opt == INDICES_32_BIT) {
    int *indice = (int*) (listIndices[idx]);
    for (int i = 0; i < total; ++i)
    {
      if (indice[i] == (int)id)
      {
        offset[idx] = i;
        break;
      }
    } 
  } else if (Opt == INDICES_64_BIT) {
    long *indice = (long*) (listIndices[idx]);
    for (int i = 0; i < total; ++i)
    {
      if (indice[i] == id)
      {
        offset[idx] = i;
        break;
      }
    } 
  } else {
    // INDICES_CPU or INDICES_IVF; no indices are being stored
  }
}

void runIVFIDInvertedListFind(long id,
                              thrust::device_vector<void*>& listIndices,
                              thrust::device_vector<int>& listLengths,
                              IndicesOptions indicesOptions,
                              Tensor<int, 1, true>& offset, // output
                              hipStream_t stream)
{
  int nlist = (int)listIndices.size();
  int numThreads = std::min(nlist, getMaxThreadsCurrentDevice());
  int numBlocks = utils::divUp(nlist, numThreads);

  dim3 grid(numBlocks);
  dim3 block(numThreads);

#define RUN_FIND(IND)                                      \
  do {                                                     \
    ivfpqInvertedListFind<IND><<<grid, block, 0, stream>>>(\
      id, \
      offset.getSize(0), \
      listIndices.data().get(), \
      listLengths.data().get(), \
      offset);     \
  } while (0)

  if ((indicesOptions == INDICES_CPU) || (indicesOptions == INDICES_IVF)) {
    // no need to maintain indices on the GPU
    RUN_FIND(INDICES_IVF);
  } else if (indicesOptions == INDICES_32_BIT) {
    RUN_FIND(INDICES_32_BIT);
  } else if (indicesOptions == INDICES_64_BIT) {
    RUN_FIND(INDICES_64_BIT);
  } else {
    // unknown index storage type
    FAISS_ASSERT(false);
  }
  CUDA_TEST_ERROR();

#undef RUN_FIND
}

// only use one thread to delete id
template <IndicesOptions Opt>
__global__ void
ivfpqInvertedListRemove(int listIdx,
                        int listPos,
                        void** listIndices,
                        int* listLengths) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx > 1) {
    return;
  }

  // use last element fit in the position you want deleted
  if (Opt == INDICES_32_BIT) {
    int *indice = (int*) listIndices[listIdx];
    indice[listPos] = indice[listLengths[listIdx] - 1];
  } else if (Opt == INDICES_64_BIT) {
    long *indice = (long *) listIndices[listIdx];
    indice[listPos] = indice[listLengths[listIdx] - 1];
  } else {
    // INDICES_CPU or INDICES_IVF; no indices are being stored
  }
}

void runIVFIDInvertedListRemove(int listIdx,
                                int listPos,
                                thrust::device_vector<void*>& listIndices,
                                thrust::device_vector<int>& listLengths,
                                IndicesOptions indicesOptions,
                                hipStream_t stream)
{
#define RUN_REMOVE(IND)                               \
  do {                                                \
    ivfpqInvertedListRemove<IND><<<1, 1, 0, stream>>>(\
      listIdx, \
      listPos, \
      listIndices.data().get(), \
      listLengths.data().get());\
  } while (0)

  if ((indicesOptions == INDICES_CPU) || (indicesOptions == INDICES_IVF)) {
    // no need to maintain indices on the GPU
    RUN_REMOVE(INDICES_IVF);
  } else if (indicesOptions == INDICES_32_BIT) {
    RUN_REMOVE(INDICES_32_BIT);
  } else if (indicesOptions == INDICES_64_BIT) {
    RUN_REMOVE(INDICES_64_BIT);
  } else {
    // unknown index storage type
    FAISS_ASSERT(false);
  }
  CUDA_TEST_ERROR();

#undef RUN_FIND
}

} } // namespace
