#include "hip/hip_runtime.h"
#include <unordered_map>
#include <iostream>
#include <sstream>

#include <thrust/copy.h>

#include "../GpuResources.h"
#include "../utils/HostTensor.cuh"
#include "../utils/CopyUtils.cuh"

#include "IVFID.cuh"
#include "InvertedListAppend.cuh"


namespace faiss { namespace gpu {

using namespace std; 
using namespace faiss::gpu;

void IVFID::getThrustVector()
{
  std::cout << "------------------------" << std::endl;
  std::cout << "deviceListPointers ---> ";
  thrust::copy(deviceListIndexPointers_.begin(),
               deviceListIndexPointers_.end(),
               std::ostream_iterator<void*>(std::cout, " "));

  std::cout << "\ndeviceListPointers ---> ";
  thrust::copy(deviceListLengths_.begin(),
               deviceListLengths_.end(),
               std::ostream_iterator<int>(std::cout, " "));
  std::cout << "-------------------------" << std::endl;
}

IVFID::IVFID(GpuResources* resources,
               /// We do not own this reference
               IndicesOptions indicesOptions,
               MemorySpace space)
    : IVFBase(resources, indicesOptions, space, NLIST) 
{
}

IVFID::~IVFID()
{
}

void IVFID::add_from_cpu(int listId,
                        const long* indices,
                        size_t numVecs) {
  // This list must already exist
  FAISS_ASSERT(listId < deviceListIndices_.size());
  auto stream = resources_->getDefaultStreamCurrentDevice();

  // If there's nothing to add, then there's nothing we have to do
  if (numVecs == 0) {
    return;
  }

  int preNum = deviceListIndices_[listId]->size() / sizeof(int);

  // Handle the indices as well
  addIndicesFromCpu_(listId, indices, numVecs);

  // And our size has changed too
  int listLength = preNum + numVecs;
  deviceListLengths_[listId] = listLength; 

  // We update this as well, since the multi-pass algorithm uses it
  maxListLength_ = std::max(maxListLength_, listLength);

  // device_vector add is potentially happening on a different stream
  // than our default stream
  if (stream != 0) {
    streamWait({stream}, {0});
  }
}

/*
 * 0. get assign count
 * 1. alloc memory in GPU
 * 2. generate tensor
 * 3. update info
 */
int IVFID::add(const vector<int>& ids)
{
    if (ids.empty()) { return 0; }

    auto& mem = resources_->getMemoryManagerCurrentDevice();
    auto stream = resources_->getDefaultStreamCurrentDevice();

    // vector id -> offset in list
    // (we already have vector id -> list id in listIds)
    HostTensor<int, 1, true> listOffsetHost({ static_cast<int>(ids.size()) });
    HostTensor<int, 1, true> listIdsHost({ static_cast<int>(ids.size()) });

    unordered_map<int, int> assignCounts;
    for (int i = 0; i < ids.size(); ++i)
    {
        int ivfIdx = ids[i] % numLists_;
        int offset = deviceListIndices_[ivfIdx]->size() / sizeof(int);

        auto it = assignCounts.find(ivfIdx);
        if (it != assignCounts.end())
        {
            offset += it->second; 
            ++it->second;
        } else {
            assignCounts[ivfIdx] = 1;
        }
        listIdsHost[i] = ivfIdx;
        listOffsetHost[i] = offset;
    }

    {
        ostringstream strIds;
        ostringstream strOffset;
        for (int i = 0; i < ids.size(); ++i)
        {
            strIds << listIdsHost[i] << " "; 
            strOffset << listOffsetHost[i] << " ";
        }
        cout << "listIds, size:" << listIdsHost.getSize(0) << ":" << strIds.str() << endl;
        cout << "listOffsetHost, size:" << listOffsetHost.getSize(0) << ":" << strOffset.str() << endl;
    }

    {
        cout << "before resize, maxListLength_:" << maxListLength_ << endl;
        for (int i = 0; i < NLIST; ++i)
        {
            auto &indices = deviceListIndices_[i];
            cout << "indices[" << i << "]'s size:" << indices->size() << " cap:" << indices->capacity() << endl;
        }
        // resize device vector
        for (const auto &count : assignCounts) 
        {
            auto &indices = deviceListIndices_[count.first];
            int newSize = count.second;
            if (indicesOptions_ == INDICES_32_BIT) {
                size_t indexSize = sizeof(int);
                newSize += indices->size() / indexSize;
                indices->resize(indices->size() + indexSize * count.second, stream);
            } else {
                FAISS_ASSERT(indicesOptions_ == INDICES_IVF);
            }
            maxListLength_ = std::max(maxListLength_, newSize);
        }
        cout << "after resize, maxListLength_:" << maxListLength_ << endl;
        for (int i = 0; i < NLIST; ++i)
        {
            auto &indices = deviceListIndices_[i];
            cout << "indices[" << i << "]'s size:" << indices->size() << " cap:" << indices->capacity() << endl;
        }

        // Update all pointers and sizes on the device for lists that we appended to
        vector<int> listIds(assignCounts.size());
        int i = 0;
        for (auto& count: assignCounts)
        {
            cout << "count.first:" << count.first << ", count.second:" << count.second << endl;
            listIds[i++] = count.first;
        }
        updateDeviceListInfo_(listIds, stream);
    }

    {
        // generate indices
        vector<long> longIdx(ids.begin(), ids.end());
        auto indices = toDevice<long, 1>(resources_,
                              0,  // gpu device idx, default 0
                              const_cast<long*>(longIdx.data()),
                              stream,
                              {(int) ids.size()});

        // host data struct to device struct
        DeviceTensor<int, 1, true> listIds(mem, listIdsHost, stream);
        DeviceTensor<int, 1, true> listOffset(mem, listOffsetHost, stream);

        cout << "listIds size:" << listIds.getSize(0)
             << ", listOffset size:" << listOffset.getSize(0)
             << ", indices size:" << indices.getSize(0) << endl;

        // Now, for each list to which a vector is being assigned, write it
        // listIDs change to device tensor
        runIVFIDInvertedListAppend(listIds,
                                   listOffset,
                                   indices,
                                   deviceListIndexPointers_,
                                   indicesOptions_,
                                   stream);
        auto checkList = getListIndices(0);
    }
}

int IVFID::remove_id(int id)
{
    auto stream = resources_->getDefaultStreamCurrentDevice();

    DeviceTensor<int, 1, true> offset({NLIST});
    // TODO
    runIVFIDInvertedListFind((long)id,
                             deviceListIndexPointers_,
                             deviceListLengths_,
                             indicesOptions_,
                             offset,
                             stream);
    int *hostOffset = new int[NLIST];
    fromDevice(offset, hostOffset, stream);
    ostringstream oss;
    oss << "offset:";
    for (int i = 0; i < NLIST; ++i)
    {
        oss << " " << hostOffset[i];
    }
    cout << oss.str() << endl;
    delete []hostOffset;
    return 0;
}

/*
 * delete one id from gpu per time.
 * 0. delete from gpu
 * 1. update info (DeviceVector, IVFID)
 */
int IVFID::remove_ids(const std::vector<int>& ids)
{
    // TODO
    return 0;
}

void IVFID::dump_ids()
{
    for (int i = 0; i < NLIST; ++i)
    {
        auto indice = getListIndices(i);
        ostringstream oss;
        oss << "list[" << i << "]: ";
        for (const auto &id : indice)
        {
            oss << id << " "; 
        }
        cout << oss.str() << endl; 
    }
}

} }